#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        std::cout << "hipGetDeviceCount failed!" << std::endl;
        return 1;
    }

    if (deviceCount == 0) {
        std::cout << "There are no available CUDA-enabled devices." << std::endl;
    } else {
        std::cout << "Detected " << deviceCount << " CUDA Capable device(s)" << std::endl;
    }

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout << "\nDevice " << dev << ": \"" << deviceProp.name << "\"" << std::endl;
    }

    return 0;
}