
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

// This wrapper function will be called from Dart.
extern "C" __declspec(dllexport)
void multiplyMatrices(float* h_a, float* h_b, float* h_c, int m, int k, int n) {
    // A is an m x k matrix
    // B is a k x n matrix
    // C is the resulting m x n matrix

    float *d_a, *d_b, *d_c;
    int a_size = m * k * sizeof(float);
    int b_size = k * n * sizeof(float);
    int c_size = m * n * sizeof(float);

    // 1. Allocate memory on the GPU device
    hipMalloc((void**)&d_a, a_size);
    hipMalloc((void**)&d_b, b_size);
    hipMalloc((void**)&d_c, c_size);

    // 2. Copy matrices from host (CPU) to device (GPU)
    hipMemcpy(d_a, h_a, a_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, b_size, hipMemcpyHostToDevice);

    // 3. Use cuBLAS to perform the matrix multiplication
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Set scaling factors for C = alpha*(A*B) + beta*C
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // The main cuBLAS call for Single-precision General Matrix Multiplication (SGEMM)
    // IMPORTANT: cuBLAS uses column-major order, while C++/Dart use row-major.
    // A standard trick is to compute C = B^T * A^T which is equivalent to C = A * B
    // in row-major. We do this by swapping A and B in the call and their dimensions.
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, m, k,
                &alpha,
                d_b, n,
                d_a, k,
                &beta,
                d_c, n);

    // Clean up the cuBLAS handle
    hipblasDestroy(handle);

    // 4. Copy the result matrix C from device (GPU) back to host (CPU)
    hipMemcpy(h_c, d_c, c_size, hipMemcpyDeviceToHost);

    // 5. Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}